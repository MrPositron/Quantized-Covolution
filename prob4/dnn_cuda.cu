#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

extern "C" {
#include "dnn_cuda.h"
}

__global__ void multiply(float *a, float *b, float*c, int m, int n, int k)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0;

    if ( col < n && row < m){
        for (int i = 0; i < k; i++){
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
        
    }
}

void matrix_mul(float *a, float *b, float *c, int m, int n, int k)
{
    float *dev_a, *dev_b, *dev_c;
    hipMalloc( (void**)&dev_a, (m*k) * sizeof(float) ) ;
    hipMalloc( (void**)&dev_b, (k*n) * sizeof(float) );
    hipMalloc( (void**)&dev_c, (m*n) * sizeof(float) );
    
    hipMemcpy( dev_a, a, (m*k) * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, (k*n) * sizeof(float), hipMemcpyHostToDevice );

    int bx = 512;
    dim3 blockDim(bx, bx);
    int gx, gy;
    if (m % bx == 0) gx = (m/bx);
    else gx = (m/bx) + 1;

    if (n % bx == 0) gy = (n/bx);
    else gy = (n/bx) + 1;
    dim3 gridDim(gx, gy);
    struct timeval stop, start;
    gettimeofday(&start, NULL);
    multiply<<<blockDim,gridDim>>>(dev_a, dev_b, dev_c, m, n, k);
    hipDeviceSynchronize();
    gettimeofday(&stop, NULL);
    printf("CUDA Inference time is %lu microseconds\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);
    hipMemcpy( c, dev_c, (m*n) * sizeof(float), hipMemcpyDeviceToHost )   ;
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}